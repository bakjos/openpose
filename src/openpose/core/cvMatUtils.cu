#include "hip/hip_runtime.h"
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include "openpose/utilities/cuda.hpp"
#include "openpose/utilities/errorAndLog.hpp"
#include "openpose/utilities/cuda.hpp"

namespace op {

	//float* (deep net format) : C x H x W
	//cv::Mat(OpenCV format) : H x W x C

	__global__ void gpuMatToFloatKernel(float* floatImagePtr, const uchar* imgData, int channels, unsigned int width, unsigned int height, int step, bool normalize, unsigned long offset) {
		float* floatImage = floatImagePtr + offset;
		const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
		const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

		if (x >= width || y >= height)
		{
			return;
		}

		
		const auto originFramePtrOffset = (y*step) + (x*channels);

		for (auto c = 0; c < channels; c++)
		{
			const auto floatImageOffsetC = c * height;
			const auto floatImageOffsetY = (floatImageOffsetC + y) * width;
			auto val = float(imgData[originFramePtrOffset + c]);
			if (normalize) {
				val = (val /256.f) - 0.5f;
			}
			floatImage[floatImageOffsetY + x] = val;
		}
		
	}

	__global__ void floatTogpuMatKernel(uchar* imgData, const float* floatImage, int channels, unsigned int width, unsigned int height, int step) {
		const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
		const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

		if (x >= width || y >= height)
		{
			return;
		}

		const auto offsetBetweenChannels = width * height;
		const auto cvMatOffset = (y*step) + (x*channels);
		for (auto c = 0; c < channels; c++)
		{
			const auto offsetChannelC = c*offsetBetweenChannels;
			const auto floatImageOffsetY = offsetChannelC + y*width;
			const auto value = uchar(__float2int_rn(floatImage[floatImageOffsetY + x]));
			imgData[cvMatOffset + c] = value;
		}

	}
	
	void gpuMatToFloatPtr(float* floatImage, const unsigned char* imgData, const int channels, const cv::Size& sourceSize, const size_t step, const bool normalize, const unsigned long offset) {
		dim3 threadsPerBlock;
		dim3 numBlocks;
		std::tie(threadsPerBlock, numBlocks) = getNumberCudaThreadsAndBlocks(sourceSize);		

		gpuMatToFloatKernel << <threadsPerBlock, numBlocks >> > (floatImage, imgData, channels, sourceSize.width, sourceSize.height, step, normalize, offset);
		cudaCheck(__LINE__, __FUNCTION__, __FILE__);
	}

	void floatPtrToGpuMat(unsigned char* imgData, const float* floatImage, const int channels, const cv::Size& sourceSize, const size_t step) {
		dim3 threadsPerBlock;
		dim3 numBlocks;
		std::tie(threadsPerBlock, numBlocks) = getNumberCudaThreadsAndBlocks(sourceSize);
		floatTogpuMatKernel << <threadsPerBlock, numBlocks >> > (imgData, floatImage, channels, sourceSize.width, sourceSize.height, step);
	}

}